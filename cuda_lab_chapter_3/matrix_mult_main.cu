
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void matrixVectorMulKernel(float *A, float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        float sum = 0.0f;
        for (int j = 0; j < N; j++) {
            sum += B[i * N + j] * C[j];
        }
        A[i] = sum;
    }
}

void matrixVectorMultiply(float *A, float *B, float *C, int N) {
    float *d_A, *d_B, *d_C;

    // Asignar memoria en el dispositivo
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // Copiar datos al dispositivo
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * sizeof(float), hipMemcpyHostToDevice);

    // Configurar y lanzar kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    matrixVectorMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copiar resultado al host
    hipMemcpy(A, d_A, N * sizeof(float), hipMemcpyDeviceToHost);

    // Liberar memoria del dispositivo
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}